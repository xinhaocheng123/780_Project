#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cute/layout.hpp>
#include <cute/tensor.hpp>
#include <cutlass/cutlass.h>
#define USE_GRACE_HOPPER
#include "copy_sm80.h"
#include "hopper_matmul.h"
#include "pipeline.h"
#include "tma_copy.h"
using namespace cute;
static constexpr  int NUM_WARPS_PER_GROUP = 4;

template <class TMA_10000003, class TMA_10000004>
__global__ void  __launch_bounds__(256) custom_kernel_0(CUTE_GRID_CONSTANT TMA_10000003 const tma_10000003, CUTE_GRID_CONSTANT TMA_10000004 const tma_10000004,  bfloat16_t* dtensor10000005_ptr, bfloat16_t const* dtensor10000003_ptr, bfloat16_t const* dtensor10000004_ptr) {
int thread_idx = threadIdx.x;
static constexpr int NUM_THREADS = 128;
static constexpr int CONSUMER_NUM_THREADS = 128;
// STensors
extern __shared__ char buf[];
bfloat16_t *stensor20000015_ptr = (bfloat16_t*)(buf + 128);
bfloat16_t *stensor20000013_ptr = (bfloat16_t*)(buf + 32896);
bfloat16_t *stensor20000012_ptr = (bfloat16_t*)(buf + 128);
// G->S copy atoms
// Copy for G->S: dtensor 10000003 -> stensor 20000012
using DTensor10000003TileLayout = Layout<Shape<Int<64>, Int<128>>, Stride<Int<1>, Int<4096>>>;
tb::HopperAsyncPipeline<2> hopper_async_pipeline_20000012((void *) (buf + 49280), (tb::warpgroup_id() == 1 && tb::warp_id() % NUM_WARPS_PER_GROUP == 0), tb::warpgroup_id() < 1, 16384, 1);
using STensor20000012InputAtom = tb::TMA_COPY<bfloat16_t, decltype(composition(Swizzle<3, 4, 3>{}, Layout<Shape<Int<128>, Int<64>>, Stride<Int<64>, Int<1>>>{})), Layout<Shape<Int<128>, Int<64>, Int<1>, Int<1>, Int<1>, Int<64>>, Stride<Int<4096>, Int<1>, Int<1>, Int<4096>, Int<1>, Int<64>>>, decltype(tma_10000003), decltype(hopper_async_pipeline_20000012), true, 64>;
// Copy for G->S: dtensor 10000004 -> stensor 20000013
using DTensor10000004TileLayout = Layout<Shape<Int<64>, Int<64>>, Stride<Int<1>, Int<4096>>>;
tb::HopperAsyncPipeline<2> hopper_async_pipeline_20000013((void *) (buf + 49312), (tb::warpgroup_id() == 1 && tb::warp_id() % NUM_WARPS_PER_GROUP == 0), tb::warpgroup_id() < 1, 8192, 1);
using STensor20000013InputAtom = tb::TMA_COPY<bfloat16_t, decltype(composition(Swizzle<3, 4, 3>{}, Layout<Shape<Int<64>, Int<64>>, Stride<Int<1>, Int<64>>>{})), Layout<Shape<Int<64>, Int<64>, Int<64>, Int<1>, Int<1>, Int<64>>, Stride<Int<1>, Int<4096>, Int<64>, Int<1>, Int<1>, Int<262144>>>, decltype(tma_10000004), decltype(hopper_async_pipeline_20000013), false, 64>;

__syncthreads();
  *((uint128_t*)buf) = 0ul;
  
  
  
  // S->G copy atoms
  // Copy for S->G: stensor 20000015 -> dtensor 10000005
  bfloat16_t *dtensor10000005_tile_ptr = dtensor10000005_ptr  + blockIdx.x*64*1;
  using DTensor10000005TileLayout = Layout<Shape<Int<64>, Int<128>>, Stride<Int<1>, Int<4096>>>;
  using STensor20000015OutputAtom = tb::OutputChunkedSyncCopy<bfloat16_t, DTensor10000005TileLayout, decltype(composition(Swizzle<3, 4, 3>{}, Layout<Shape<Int<64>, Int<128>>, Stride<Int<1>, Int<64>>>{})), NUM_THREADS>;
  
  
  using Matmul20000015LayoutA = decltype(composition(Swizzle<3, 4, 3>{}, Layout<Shape<Int<64>, Int<128>>, Stride<Int<1>, Int<64>>>{}));
  using Matmul20000015LayoutB = decltype(composition(Swizzle<3, 4, 3>{}, Layout<Shape<Int<64>, Int<64>>, Stride<Int<1>, Int<64>>>{}));
  using Matmul20000015LayoutC = decltype(composition(Swizzle<3, 4, 3>{}, Layout<Shape<Int<64>, Int<128>>, Stride<Int<1>, Int<64>>>{}));
  using Matmul20000015Kernel = tb::Hopper_Matmul<bfloat16_t, true, false, Matmul20000015LayoutA, Matmul20000015LayoutB, Matmul20000015LayoutC, NUM_THREADS, 0, false, false, true, true, 2>;
  auto matmul_20000015_accum = Matmul20000015Kernel::get_mma_rC(thread_idx);
  
  __syncthreads();
  int warpgroup_id = tb::warpgroup_id();
  if (warpgroup_id == 1) {
    if (tb::warp_id_in_wg() == 0) {
      for (uint32_t for_idx = 0; for_idx < 64; for_idx++) {
        STensor20000012InputAtom::run(tma_10000003, stensor20000012_ptr,  -1, 0, -1, for_idx, hopper_async_pipeline_20000012);
        STensor20000013InputAtom::run(tma_10000004, stensor20000013_ptr,  1, -1, -1, for_idx, hopper_async_pipeline_20000013);
      }
    }
  }
  else {
    // Consumer main loop
    for (uint32_t for_idx = 0; for_idx < 64; for_idx++) {
      {
        // OP type: tb_matmul_op
        int read_idx_20000012 = hopper_async_pipeline_20000012.consumer_wait();
        int read_idx_20000013 = hopper_async_pipeline_20000013.consumer_wait();
        Matmul20000015Kernel::run(matmul_20000015_accum, stensor20000012_ptr, stensor20000013_ptr, (char*)(buf+0), thread_idx, read_idx_20000012);
        tb::wg_sync<CONSUMER_NUM_THREADS>(8);
      }
      hopper_async_pipeline_20000012.consumer_release();
      hopper_async_pipeline_20000013.consumer_release();
    }
    // Write back in-register accumulators
    tb::wg_sync<CONSUMER_NUM_THREADS>(8);
    Matmul20000015Kernel::write_back_mma_rC(stensor20000015_ptr, matmul_20000015_accum, thread_idx);
    // The epilogue (kernels outside the loop)
    tb::wg_sync<CONSUMER_NUM_THREADS>(8);
    {
      // OP type: tb_output_op
      STensor20000015OutputAtom::run(dtensor10000005_tile_ptr, stensor20000015_ptr, thread_idx);
    }
  }
}




 void gemm_kernel_bf16(std::vector<void const *> input_tensors, std::vector<void*> output_tensors){
  {
    // OP type: kn_input_op
  }
  {
    // OP type: kn_input_op
  }
  
  {
    // OP type: kn_customized_op
    bfloat16_t *dtensor10000005 = (bfloat16_t*)output_tensors.at(0);
    bfloat16_t *dtensor10000003 = (bfloat16_t*)input_tensors.at(0);
    bfloat16_t *dtensor10000004 = (bfloat16_t*)input_tensors.at(1);
    dim3 grid_dim(64, 1, 1);
    dim3 block_dim(256, 1, 1);
    size_t smem_size = 49344;
    
    // define tmas
    std::vector<bool> minputs = {true, false};
    static constexpr cute::GMMA::Major GmmaMajor_10000003 = GMMA::Major::K;
    using DstMNKLayout_10000003 = decltype(composition(Swizzle<3, 4, 3>{}, Layout<Shape<Int<128>, Int<64>>, Stride<Int<64>, Int<1>>>{}));
    using SrcMNKLayout_10000003 = Layout<Shape<Int<128>, Int<64>, Int<1>, Int<1>, Int<1>, Int<64>>, Stride<Int<4096>, Int<1>, Int<1>, Int<4096>, Int<1>, Int<64>>>;
    using SmemLayoutAtom_10000003 = decltype(cutlass::gemm::collective::detail::ss_smem_selector<GmmaMajor_10000003, bfloat16_t, decltype(get<0>(DstMNKLayout_10000003{})), decltype(get<1>(DstMNKLayout_10000003{}))>());
    using DstPipeLayout_10000003 = decltype(tile_to_shape(SmemLayoutAtom_10000003{}, make_shape(shape<0>(DstMNKLayout_10000003{}), shape<1>(DstMNKLayout_10000003{}), Int<2>{}), Step<_1, _2, _3>{}));
    auto g_tensor_10000003 = make_tensor(make_gmem_ptr<bfloat16_t>(dtensor10000003), SrcMNKLayout_10000003{});
    auto tma_10000003 = make_tma_copy(SM90_TMA_LOAD{}, g_tensor_10000003, DstPipeLayout_10000003{}(_, _, Int<0>{}));
    
    static constexpr cute::GMMA::Major GmmaMajor_10000004 = GMMA::Major::MN;
    using DstMNKLayout_10000004 = decltype(composition(Swizzle<3, 4, 3>{}, Layout<Shape<Int<64>, Int<64>>, Stride<Int<1>, Int<64>>>{}));
    using SrcMNKLayout_10000004 = Layout<Shape<Int<64>, Int<64>, Int<64>, Int<1>, Int<1>, Int<64>>, Stride<Int<1>, Int<4096>, Int<64>, Int<1>, Int<1>, Int<262144>>>;
    using SmemLayoutAtom_10000004 = decltype(cutlass::gemm::collective::detail::ss_smem_selector<GmmaMajor_10000004, bfloat16_t, decltype(get<0>(DstMNKLayout_10000004{})), decltype(get<1>(DstMNKLayout_10000004{}))>());
    using DstPipeLayout_10000004 = decltype(tile_to_shape(SmemLayoutAtom_10000004{}, make_shape(shape<0>(DstMNKLayout_10000004{}), shape<1>(DstMNKLayout_10000004{}), Int<2>{}), Step<_1, _2, _3>{}));
    auto g_tensor_10000004 = make_tensor(make_gmem_ptr<bfloat16_t>(dtensor10000004), SrcMNKLayout_10000004{});
    auto tma_10000004 = make_tma_copy(SM90_TMA_LOAD{}, g_tensor_10000004, DstPipeLayout_10000004{}(_, _, Int<0>{}));
    
    hipFuncSetAttribute(custom_kernel_0<decltype(tma_10000003), decltype(tma_10000004)>, hipFuncAttributeMaxDynamicSharedMemorySize, 49344);
    custom_kernel_0<<<grid_dim, block_dim, smem_size>>>(tma_10000003, tma_10000004,  dtensor10000005, dtensor10000003, dtensor10000004);
    hipDeviceSynchronize();
  }
  {
    // OP type: kn_output_op
  }
}